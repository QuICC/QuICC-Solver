#include "hip/hip_runtime.h"
#include <cassert>
#include <complex>
#include <cuda/std/complex>

#include "Cuda/CudaUtil.hpp"
#include "Pointwise.hpp"
#include "ViewOps/Pointwise/Functors.hpp"

namespace QuICC {
namespace Pointwise {
namespace Cuda {

namespace details {
// naive implementation
template <class Functor, class Tout, class ...Targs>
__global__ void pointwiseKernel(Functor f, Tout out, Targs... args)
{
   const auto M = out.size();
   const std::size_t m = blockIdx.x * blockDim.x + threadIdx.x;
   if (m < M)
   {
      out[m] = f(args[m]...);
   }
}

} // namespace details


template <class Functor, class Tout, class ...Targs>
void Op<Functor, Tout, Targs...>::applyImpl(Tout& out, const Targs&... args)
{
   Profiler::RegionFixture<4> fix("Pointwise::Cuda::applyImpl");

   assert(QuICC::Cuda::isDeviceMemory(out.data()));
   assert(((QuICC::Cuda::isDeviceMemory(args.data())) && ...));

   // check Tout and Targs.. match Functor op
   using res_t = std::invoke_result_t<Functor, typename Targs::ScalarType...>;
   static_assert(std::is_same_v<typename Tout::ScalarType, res_t>,
      "Mismatch in functor or arguments");
   // check same size
   assert(((out.size() == args.size()) && ... ));

   const auto M = out.size();
   dim3 blockSize;
   dim3 numBlocks;
   blockSize.x = 64;
   numBlocks.x = (M + blockSize.x - 1) / blockSize.x;

   details::pointwiseKernel<Functor, Tout, Targs...>
      <<<numBlocks, blockSize>>>(_f, out, args...);
}


// Explicit instantiations
// tests
template class Op<SquareFunctor<double>, View::ViewBase<double>, View::ViewBase<double>>;
template class Op<Abs2Functor<double>, View::ViewBase<double>, View::ViewBase<std::complex<double>>>;
// JW
template class Op<Abs2Functor<double>, View::View<double, View::DCCSC3DJIK>,
   View::View<std::complex<double>, View::DCCSC3DJIK>>;
// Add Mods
template class Op<AddFunctor<cuda::std::complex<double>>,
   View::ViewBase<cuda::std::complex<double>>,
   View::ViewBase<cuda::std::complex<double>>,
   View::ViewBase<cuda::std::complex<double>>>;
template class Op<AddFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::DCCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::DCCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::DCCSC3DJIK>>;
template class Op<AddFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::DCCSC3D>,
   View::View<cuda::std::complex<double>, View::DCCSC3D>,
   View::View<cuda::std::complex<double>, View::DCCSC3D>>;
template class Op<AddFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3DJIK>>;
template class Op<AddFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3D>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3D>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3D>>;
template class Op<AddFunctor<double>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>>;
// Sub Mods
template class Op<SubFunctor<cuda::std::complex<double>>,
   View::ViewBase<cuda::std::complex<double>>,
   View::ViewBase<cuda::std::complex<double>>,
   View::ViewBase<cuda::std::complex<double>>>;
template class Op<SubFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::DCCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::DCCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::DCCSC3DJIK>>;
template class Op<SubFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::DCCSC3D>,
   View::View<cuda::std::complex<double>, View::DCCSC3D>,
   View::View<cuda::std::complex<double>, View::DCCSC3D>>;
template class Op<SubFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3DJIK>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3DJIK>>;
template class Op<SubFunctor<cuda::std::complex<double>>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3D>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3D>,
   View::View<cuda::std::complex<double>, View::S1CLCSC3D>>;
template class Op<SubFunctor<double>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>>;
// Dot
template class Op<DotFunctor<double>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>>;
// Cross
template class Op<CrossCompFunctor<double>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>,
   View::View<double, View::DCCSC3D>>;

} // namespace Cuda
} // namespace Pointwise
} // namespace QuICC

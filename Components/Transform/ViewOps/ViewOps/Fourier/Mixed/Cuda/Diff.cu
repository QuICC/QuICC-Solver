#include "hip/hip_runtime.h"
#include <complex>
#include <hip/hip_complex.h>
#include <iostream>

#include "Diff.hpp"
#include "View/View.hpp"
#include "ViewOps/Fourier/Util.hpp"
#include "ViewOps/Fourier/Tags.hpp"
#include "Cuda/CudaUtil.hpp"
#include "Profiler/Interface.hpp"

namespace QuICC {
namespace Transform {
namespace Fourier {
namespace Mixed {
namespace Cuda {

using namespace QuICC::Memory;

using mods_t = View<std::complex<double>, DCCSC3D>;

/// @brief thread coarsening factor
constexpr std::size_t tCF = 8;

namespace details
{
    using namespace QuICC::Transform::Fourier::details;

    /// Cuda kernel
    template<std::size_t Order, class Direction, class Treatment>
    __global__ void diffKernel(mods_t out, const mods_t in, const double scale)
    {
        constexpr bool isComplex = Order % 2;
        constexpr int sgn = 1 - 2*static_cast<int>((Order/2) % 2);

        const auto M = out.dims()[0];
        const auto N = out.dims()[1];

        double fftScaling = 1.0;
        if constexpr (std::is_same_v<Direction, fwd_t>)
        {
            fftScaling = 1.0 / static_cast<double>((M-1)*2);
        }

        hipDoubleComplex c;
        if constexpr (isComplex)
        {
            c = {0.0, fftScaling};
        }
        else
        {
            c = {fftScaling, 0.0};
        }

        // map x block and thread to m loop
        const std::size_t m = blockIdx.x * blockDim.x + threadIdx.x;

        // Column major
        // Get total number of columns to loop over
        auto indices = in.indices()[1];
        auto columns = indices.size();
        if(m < M)
        {
            hipDoubleComplex tmpR = {static_cast<double>(sgn) *
                fast_pow<Order>(static_cast<double>(m)*scale), 0.0};

            auto tmpC = hipCmul(c, tmpR);

            if constexpr (std::is_same_v<Treatment, zeroP_t>)
            {
                if (m == 0)
                {
                    tmpC = {fftScaling, 0.0};
                }
            }

            if constexpr (std::is_same_v<Treatment, zeroMinusP_t>)
            {
                if (m == 0)
                {
                    tmpC = {-fftScaling, 0.0};
                }
            }

            // map y blocks to columns loop with thread coarsening
            #pragma unroll
            for (std::size_t nn = 0; nn < tCF; ++nn)
            {
                auto n = blockIdx.y * tCF + nn;
                if (n < columns)
                {
                    // linear index (m,n,k)
                    auto index = m + n*M;
                    reinterpret_cast<hipDoubleComplex*>(out.data())[index] =
                        hipCmul(reinterpret_cast<hipDoubleComplex*>(in.data())[index], tmpC);
                }
            }
        }
    }

}

template<class Tout, class Tin, std::size_t Order, class Direction, class Treatment>
DiffOp<Tout, Tin, Order, Direction, Treatment>::DiffOp(ScaleType scale) : mScale(scale){};

template<class Tout, class Tin, std::size_t Order, class Direction, class Treatment>
void DiffOp<Tout, Tin, Order, Direction, Treatment>::applyImpl(Tout& out, const Tin& in)
{
    Profiler::RegionFixture<4> fix("DiffOp::applyImpl");

    assert(out.dims()[0] == in.dims()[0]);
    assert(out.dims()[1] == in.dims()[1]);
    assert(out.dims()[2] == in.dims()[2]);
    assert(QuICC::Cuda::isDeviceMemory(out.data()));

    if constexpr (std::is_same_v<Direction, bwd_t> &&
        std::is_same_v<Treatment, void> &&  Order == 0)
    {
        // if the diff is in place it is a noop
        if(out.data() == in.data())
        {
            return;
        }
    }

    dim3 blockSize;
    blockSize.x = 64;
    blockSize.y = 1;
    blockSize.z = 1;
    dim3 numBlocks;
    numBlocks.x = (in.dims()[0] + blockSize.x - 1) / blockSize.x;
    auto indices = in.indices()[1];
    auto columns = indices.size();
    numBlocks.y = (columns + tCF - 1) / tCF;
    numBlocks.z = 1;
    details::diffKernel<Order, Direction, Treatment><<<numBlocks, blockSize>>>(out, in, mScale);
}

// explicit instantations
template class DiffOp<mods_t, mods_t, 0, fwd_t>;
template class DiffOp<mods_t, mods_t, 1, fwd_t>;
template class DiffOp<mods_t, mods_t, 1, fwd_t, zeroP_t>;
template class DiffOp<mods_t, mods_t, 1, fwd_t, zeroMinusP_t>;
template class DiffOp<mods_t, mods_t, 2, fwd_t>;
template class DiffOp<mods_t, mods_t, 3, fwd_t>;
template class DiffOp<mods_t, mods_t, 4, fwd_t>;
template class DiffOp<mods_t, mods_t, 0, bwd_t>;
template class DiffOp<mods_t, mods_t, 1, bwd_t>;
template class DiffOp<mods_t, mods_t, 2, bwd_t>;
template class DiffOp<mods_t, mods_t, 3, bwd_t>;
template class DiffOp<mods_t, mods_t, 4, bwd_t>;

} // namespace Cuda
} // namespace Mixed
} // namespace Fourier
} // namespace Transform
} // namespace QuICC

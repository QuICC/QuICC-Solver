#include "hip/hip_runtime.h"
// External includes
//
#include <complex>
#include <hip/hip_complex.h>
#include <iostream>

// Project includes
//
#include "Diff.hpp"
#include "View/View.hpp"
#include "ViewOps/Fourier/Util.hpp"
#include "ViewOps/Fourier/Tags.hpp"
#include "Cuda/CudaUtil.hpp"
#include "Profiler/Interface.hpp"

namespace QuICC {
namespace Transform {
namespace Fourier {
namespace Complex {
namespace Cuda {

using namespace QuICC::Memory;

using mods_t = View<std::complex<double>, DCCSC3DInOrder>;

/// @brief thread coarsening factor
constexpr std::size_t tCF = 8;

namespace details
{
    using namespace QuICC::Transform::Fourier::details;

    /// Cuda kernel
    template<std::size_t Order, class Direction, std::uint16_t Treatment>
    __global__ void diffKernel(mods_t out, const mods_t in, const double scale)
    {

        constexpr bool isComplex = Order % 2;
        constexpr int sgn = 1 - 2*static_cast<int>((Order/2) % 2);

        // dealias bounds
        const auto M = in.lds();
        const auto MDealias = in.dims()[0];
        const auto N = in.dims()[1];

        // positive / negative coeff bounds
        const auto negM = M / 2;
        const auto posM = negM + M % 2;
        const auto negDealias = MDealias / 2;
        const auto posDealias = negDealias + MDealias % 2;

        double fftScaling = 1.0;
        if constexpr (std::is_same_v<Direction, fwd_t>)
        {
            fftScaling = 1.0 / static_cast<double>(M);
        }
        hipDoubleComplex c;
        if constexpr (isComplex)
        {
            c = {0.0, fftScaling};
        }
        else
        {
            c = {fftScaling, 0.0};
        }

        const std::size_t m = blockIdx.x * blockDim.x + threadIdx.x;

        // Column major
        // Get number of columns that have k == 0
        auto pointers = in.pointers()[1];
        auto nColumnsK0 = pointers[1] - pointers[0];
        // Get total number of columns to loop over
        auto indices = in.indices()[1];
        auto columns = indices.size();
        if(m < M)
        {
            hipDoubleComplex tmpR;
            if(m < posM)
            {
                tmpR = {static_cast<double>(sgn) *
                    fast_pow<Order>(static_cast<double>(m)*scale), 0.0};
            }
            else
            {
                tmpR = {static_cast<double>(sgn) *
                    fast_pow<Order>(-static_cast<double>(M-m)*scale), 0.0};
            }

            // dealias
            if(m >= posDealias && m < M - negDealias)
            {
                tmpR = {0.0, 0.0};
            }

            hipDoubleComplex tmpC;
            if constexpr (Treatment == none_m)
            {
                tmpC = hipCmul(c, tmpR);
            }

            // map y blocks to columns loop with thread coarsening
            #pragma unroll
            for (std::size_t nn = 0; nn < tCF; ++nn)
            {
                auto n = blockIdx.y * tCF + nn;
                if (n < columns)
                {
                    if constexpr (Treatment & zeroP_m)
                    {
                        if (n < nColumnsK0 && (m == 0 || m == posM))
                        {
                            tmpC = {fftScaling, 0.0};
                        }
                        else
                        {
                            tmpC = hipCmul(c, tmpR);
                        }
                    }

                    if constexpr (Treatment & zeroMinusP_m)
                    {
                        if (n < nColumnsK0 && (m == 0 || m == posM))
                        {
                            tmpC = {-fftScaling, 0.0};
                        }
                        else
                        {
                            tmpC = hipCmul(c, tmpR);
                        }
                    }

                    if constexpr (Treatment & zeroResetMean_m)
                    {
                        if (n < nColumnsK0 && (m != 0 && m != posM))
                        {
                            tmpC = {0.0, 0.0};
                        }
                        else
                        {
                            tmpC = hipCmul(c, tmpR);
                        }
                    }

                    // linear index (m,n,k) , variable n is really # of cols
                    auto index = m + n*M;
                    reinterpret_cast<hipDoubleComplex*>(out.data())[index] =
                        hipCmul(reinterpret_cast<hipDoubleComplex*>(in.data())[index], tmpC);
                }
            }
        }
    }

}

template<class Tout, class Tin, std::size_t Order, class Direction, std::uint16_t Treatment>
DiffOp<Tout, Tin, Order, Direction, Treatment>::DiffOp(ScaleType scale) : mScale(scale){};

template<class Tout, class Tin, std::size_t Order, class Direction, std::uint16_t Treatment>
void DiffOp<Tout, Tin, Order, Direction, Treatment>::applyImpl(Tout& out, const Tin& in)
{
    Profiler::RegionFixture<4> fix("DiffOp::applyImpl");

    assert(out.size() == in.size());
    assert(out.dims()[0] == in.dims()[0]);
    assert(out.dims()[1] == in.dims()[1]);
    assert(out.dims()[2] == in.dims()[2]);
    assert(QuICC::Cuda::isDeviceMemory(out.data()));

    if constexpr (std::is_same_v<Direction, bwd_t> &&
        Treatment == none_m && Order == 0)
    {
        // if the diff is in place it is a noop
        if(out.data() == in.data() && out.dims()[0] == out.lds())
        {
            return;
        }
    }

    dim3 blockSize;
    blockSize.x = 64;
    blockSize.y = 1;
    blockSize.z = 1;
    dim3 numBlocks;
    numBlocks.x = (in.lds() + blockSize.x - 1) / blockSize.x;
    auto indices = in.indices()[1];
    auto columns = indices.size();
    numBlocks.y = (columns + tCF - 1) / tCF;
    numBlocks.z = 1;
    details::diffKernel<Order, Direction, Treatment><<<numBlocks, blockSize>>>(out, in, mScale);
}

// explicit instantations
template class DiffOp<mods_t, mods_t, 0, fwd_t>;
template class DiffOp<mods_t, mods_t, 0, fwd_t, zeroResetMean_m>;
template class DiffOp<mods_t, mods_t, 1, fwd_t>;
template class DiffOp<mods_t, mods_t, 1, fwd_t, zeroP_m>;
template class DiffOp<mods_t, mods_t, 1, fwd_t, zeroMinusP_m>;
template class DiffOp<mods_t, mods_t, 2, fwd_t>;
template class DiffOp<mods_t, mods_t, 3, fwd_t>;
template class DiffOp<mods_t, mods_t, 4, fwd_t>;
template class DiffOp<mods_t, mods_t, 0, bwd_t>;
template class DiffOp<mods_t, mods_t, 1, bwd_t>;
template class DiffOp<mods_t, mods_t, 2, bwd_t>;
template class DiffOp<mods_t, mods_t, 3, bwd_t>;
template class DiffOp<mods_t, mods_t, 4, bwd_t>;

} // namespace Cuda
} // namespace Complex
} // namespace Fourier
} // namespace Transform
} // namespace QuICC

#include "hip/hip_runtime.h"
/**
 * @file Op.hpp
 * @brief Transpose operations on Views
 */
#pragma once

// External includes
//

// Project includes
//
#include "ViewOps/Transpose/Cuda/Packing.hpp"
#include "View/View.hpp"
#include "Cuda/CudaUtil.hpp"

namespace QuICC {
/// @brief namespace for Transpose type operations
namespace Transpose {
/// @brief namespace for Cuda backends
namespace Cuda {


namespace details
{
template<class TDATA>
__global__ void pack(View::ViewBase<TDATA> buffer, const TDATA* in,
   const View::ViewBase<int> sendCountsView,
   const View::View<int, View::Attributes<View::dense2D>> sendDisplsView,
   const View::ViewBase<int> sendBufferDisplsView)
{

   const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
   const std::size_t j = blockIdx.y * blockDim.y + threadIdx.y;

   const auto I = sendDisplsView.dims()[0];
   const auto J = sendDisplsView.dims()[1];

   if (i < I && j < sendCountsView[i])
   {
      buffer[sendBufferDisplsView[i]+j] = *(in + sendDisplsView[i*J+j]);
   }
}

} // namespace details


template <class TDATA>
void pack(View::ViewBase<TDATA> buffer, const TDATA* in,
   const View::ViewBase<int> sendCountsView,
   const View::View<int, View::Attributes<View::dense2D>> sendDisplsView,
   const View::ViewBase<int> sendBufferDisplsView)
{

   const auto I = sendDisplsView.dims()[0];
   const auto J = sendDisplsView.dims()[1];

   // setup grid
   dim3 blockSize;
   dim3 numBlocks;

   blockSize.x = 16;
   blockSize.y = 64;
   blockSize.z = 1;
   numBlocks.x = (I + blockSize.x - 1) / blockSize.x;
   numBlocks.y = (J + blockSize.y - 1) / blockSize.y;
   numBlocks.z = 1;

   details::pack<TDATA>
      <<<numBlocks, blockSize>>>(buffer, in , sendCountsView, sendDisplsView, sendBufferDisplsView);

}


namespace details
{
template<class TDATA>
__global__ void unPack(TDATA* out, const View::ViewBase<TDATA> buffer,
   const View::ViewBase<int> recvCountsView,
   const View::View<int, View::Attributes<View::dense2D>> recvDisplsView,
   const View::ViewBase<int> recvBufferDisplsView)
{

   const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
   const std::size_t j = blockIdx.y * blockDim.y + threadIdx.y;

   const auto I = recvDisplsView.dims()[0];
   const auto J = recvDisplsView.dims()[1];

   if (i < I && j < recvCountsView[i])
   {
      *(out + recvDisplsView[i*J+j]) = buffer[recvBufferDisplsView[i]+j];
   }
}

} // namespace details


template <class TDATA>
void unPack(TDATA* out, const View::ViewBase<TDATA> buffer,
   const View::ViewBase<int> recvCountsView,
   const View::View<int, View::Attributes<View::dense2D>> recvDisplsView,
   const View::ViewBase<int> recvBufferDisplsView)
{

   const auto I = recvDisplsView.dims()[0];
   const auto J = recvDisplsView.dims()[1];

   // setup grid
   dim3 blockSize;
   dim3 numBlocks;

   blockSize.x = 16;
   blockSize.y = 64;
   blockSize.z = 1;
   numBlocks.x = (I + blockSize.x - 1) / blockSize.x;
   numBlocks.y = (J + blockSize.y - 1) / blockSize.y;
   numBlocks.z = 1;

   details::pack<TDATA>
      <<<numBlocks, blockSize>>>(out, buffer, recvCountsView, recvDisplsView, recvBufferDisplsView);

}



} // namespace Cuda
} // namespace Transpose
} // namespace QuICC

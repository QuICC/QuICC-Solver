/**
 * @file GpuMatrix.cu
 * @brief Simple Matrix representation for GPU data
 */

// System includes
//
#include <stdexcept>
#include <utility>

// External includes
//
#include <hipblas.h>

// Class include
//
#include "QuICC/Transform/Fft/Backend/CuFft/GpuMatrix.hpp"

// Project includes
//
#include "QuICC/Transform/Fft/Backend/CuFft/CheckCuda.hpp"

namespace QuICC {

namespace Transform {

namespace Fft {

namespace Backend {

namespace CuFft {

   GpuMatrix::GpuMatrix()
      : mData(nullptr), mMemSize(0), mRows(0), mCols(0)
   {
   }

   GpuMatrix::GpuMatrix(const int rows, const int cols)
      : mData(nullptr), mMemSize(0), mRows(rows), mCols(cols)
   {
      this->allocate(this->mRows, this->mCols);
   }

   GpuMatrix::GpuMatrix(const GpuMatrix& o)
      : mData(nullptr), mMemSize(o.mMemSize), mRows(o.mRows), mCols(o.mCols)
   {
      this->allocate(this->mRows, this->mCols);
   }

   GpuMatrix::GpuMatrix(GpuMatrix&& o)
      : mData(std::exchange(o.mData,nullptr)), mMemSize(std::exchange(o.mMemSize,0)), mRows(std::exchange(o.mRows,0)), mCols(std::exchange(o.mCols,0))
   {
      this->allocate(this->mRows, this->mCols);
   }

   GpuMatrix& GpuMatrix::operator=(const GpuMatrix& o)
   {
      if(this != &o)
      {
         if(this->size() != o.size())
         {
            this->allocate(o.mRows, o.mCols);
         } else
         {
            if(this->mRows != o.mRows)
            {
               this->mRows = o.mRows;
               this->mCols = o.mCols;
            }
         }
         hipblasHandle_t handle;
         CheckCuda(hipblasCreate(&handle), __LINE__);
         CheckCuda(hipblasDcopy(handle, this->size(), o.mData, 1, this->mData, 1), __LINE__);
         CheckCuda(hipblasDestroy(handle), __LINE__);
      }

      return *this;
   }

   GpuMatrix& GpuMatrix::operator=(GpuMatrix&& o)
   {
      if(this != &o)
      {
         this->free();
         this->mData = std::exchange(o.mData, nullptr);
         this->mMemSize = std::exchange(o.mMemSize, 0);
         this->mRows = std::exchange(o.mRows, 0);
         this->mCols = std::exchange(o.mCols, 0);
      }

      return *this;
   }

   GpuMatrix::~GpuMatrix()
   {
      this->free();
   }

   double* GpuMatrix::data()
   {
      return this->mData;
   }

   const double* GpuMatrix::data() const
   {
      return this->mData;
   }

   int GpuMatrix::rows() const
   {
      return this->mRows;
   }

   int GpuMatrix::cols() const
   {
      return this->mCols;
   }

   int GpuMatrix::size() const
   {
      return this->mRows*this->mCols;
   }

   void GpuMatrix::reshape(int rows, int cols)
   {
      if(rows == GpuMatrix::KEEP_SIZE)
      {
         rows = this->mRows;
      }

      if(cols == GpuMatrix::KEEP_SIZE)
      {
         cols = this->mCols;
      }

      if(this->mMemSize >= rows*cols)
      {
         this->mRows = rows;
         this->mCols = cols;
      } else
      {
         throw std::logic_error("Tried to resize GpuMatrix with too little memory allocated!");
      }
   }

   void GpuMatrix::resize(int rows, int cols)
   {
      if(rows == GpuMatrix::KEEP_SIZE)
      {
         rows = this->mRows;
      }

      if(cols == GpuMatrix::KEEP_SIZE)
      {
         cols = this->mCols;
      }

      this->mRows = rows;
      this->mCols = cols;

      if(this->mMemSize < rows*cols)
      {
         this->allocate(rows,cols);
      }
   }

   void GpuMatrix::allocate(const int rows, const int cols)
   {
      this->free();
      this->mMemSize = rows*cols;
      CheckCuda(hipMalloc((void**)&(this->mData), sizeof(double)*this->mMemSize), __LINE__);
      this->mRows = rows;
      this->mCols = cols;
   }

   void GpuMatrix::free()
   {
      if(this->mData)
      {
         CheckCuda(hipFree(this->mData), __LINE__);
      }
   }

}
}
}
}
}
